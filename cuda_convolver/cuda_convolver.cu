#include <new>
#include <iostream>

#include "cuda_convolver.cuh"
#include "convolve_kernel.cuh"

#include "cuda_check.h"

namespace CudaImgProc
{
    CudaConvolver::CudaConvolver(std::shared_ptr<StbImage::Image> image) : _image(image)
    {
        hipError_t cudaStatus = AllocCudaMem();
        if (cudaStatus != hipSuccess) {
            throw std::exception("Could not allocate memory for CUDA arrays!");
        }
    }

    CudaConvolver::~CudaConvolver()
    {
        DeleteCudaMem();
    }

    StbImage::Image CudaConvolver::Convolve(std::vector<float>& kernel, CudaUtil::Dim2 kernelDim)
    {
        // Calculate the number of CUDA blocks needed.
		const dim3 numBlocks(
            IntDivUp(_image->width(), blockSize.x), 
            IntDivUp(_image->height(), blockSize.y)
            );

        const size_t arrayPitch = _image->width() * _image->channels();

        // Copy the input image data to the device.
        CUDA_CHECK(hipMemcpy2DToArray(
            _devInputArray, 
            0, 
            0, 
            _image->data,
            arrayPitch,
            arrayPitch,
            _image->height(), 
            hipMemcpyHostToDevice
            ));

        // Call the convolve CUDA kernel.
        CudaUtil::Dim2 imgDim = { _image->width(), _image->height() };
        CudaUtil::Dim2 kernelRadius = { (kernelDim.x / 2), (kernelDim.y / 2) };
        CudaUtil::Dim2 smemDim = { (blockSize.x + (2 * kernelRadius.x)), (blockSize.y + (2 * kernelRadius.y)) };
        size_t smemArraySize = smemDim.x * smemDim.y * sizeof(uint32_t);
        
        ConvolveKernels::ConvolveRgba<<<numBlocks, blockSize, smemArraySize>>>(
            _devInputSurface,
            _devOutputSurface,
            imgDim,
            kernel.data(),
            kernelDim,
            smemDim
        );

        CUDA_CHECK(hipDeviceSynchronize());

        // Create a blank image with the same parameters as the source image.
        StbImage::Image output(_image->width(), _image->height(), _image->channels());

        // Copy the image data from the GPU to the output image.
        CUDA_CHECK(hipMemcpy2DFromArray(
            output.data,
            arrayPitch,
            _devOutputArray,
            0,
            0,
            arrayPitch,
            _image->height(),
            hipMemcpyDeviceToHost
            ));

        return output;
    }

    hipError_t CudaConvolver::AllocCudaMem()
    {
        hipError_t cudaStatus;

        hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);

        // Allocate the CUDA arrays for input and output.
		cudaStatus = hipMallocArray(&_devInputArray,
			&channelDesc,
			_image->width(),
			_image->height(),
			hipArraySurfaceLoadStore);
		if (cudaStatus != hipSuccess) { return cudaStatus; }

        cudaStatus = hipMallocArray(&_devOutputArray,
			&channelDesc,
			_image->width(),
			_image->height(),
			hipArraySurfaceLoadStore);
		if (cudaStatus != hipSuccess) { return cudaStatus; }

        // Create the CUDA surfaces.
        struct hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypeArray;

		resDesc.res.array.array = _devInputArray;
		cudaStatus = hipCreateSurfaceObject(&_devInputSurface, &resDesc);
		if (cudaStatus != hipSuccess) { return cudaStatus; }

        resDesc.res.array.array = _devOutputArray;
		cudaStatus = hipCreateSurfaceObject(&_devOutputSurface, &resDesc);
		if (cudaStatus != hipSuccess) { return cudaStatus; }

        return hipSuccess;
    }

    void CudaConvolver::DeleteCudaMem()
    {
        hipFreeArray(_devInputArray);
        hipFreeArray(_devOutputArray);
        hipDestroySurfaceObject(_devInputSurface);
        hipDestroySurfaceObject(_devOutputSurface);
    }
}